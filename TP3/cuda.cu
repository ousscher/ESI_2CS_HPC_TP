#include "hip/hip_runtime.h"
%%cuda
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include <hip/hip_complex.h>

#define PI 3.14159265358979323846
#define BLOCK_SIZE 256

typedef struct {
    double real;
    double imag;
} ComplexNumber;

// Kernel CUDA pour le calcul de la TFD
__global__ void dftKernel(double* signal_real, double* signal_imag, 
                         hipDoubleComplex* result, int N) {
    int k = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (k < N) {
        hipDoubleComplex sum = make_hipDoubleComplex(0.0, 0.0);
        
        for (int n = 0; n < N; n++) {
            double angle = 2 * PI * k * n / N;
            double cos_angle = cos(angle);
            double sin_angle = sin(angle);
            
            // Multiplication complexe
            double real_part = signal_real[n] * cos_angle + signal_imag[n] * sin_angle;
            double imag_part = signal_imag[n] * cos_angle - signal_real[n] * sin_angle;
            
            sum = hipCadd(sum, make_hipDoubleComplex(real_part, -imag_part));
        }
        
        result[k] = sum;
    }
}

void checkCudaError(hipError_t error, const char *message) {
    if (error != hipSuccess) {
        fprintf(stderr, "CUDA Error: %s - %s\n", message, hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }
}

void computeDFT_CUDA(ComplexNumber* signal, int N, ComplexNumber* result) {
    double *d_signal_real, *d_signal_imag;
    hipDoubleComplex *d_result;
    
    double* signal_real = (double*)malloc(N * sizeof(double));
    double* signal_imag = (double*)malloc(N * sizeof(double));
    
    for (int i = 0; i < N; i++) {
        signal_real[i] = signal[i].real;
        signal_imag[i] = signal[i].imag;
    }
    checkCudaError(hipMalloc((void**)&d_signal_real, N * sizeof(double)), 
                  "Allocation signal réel");
    checkCudaError(hipMalloc((void**)&d_signal_imag, N * sizeof(double)), 
                  "Allocation signal imaginaire");
    checkCudaError(hipMalloc((void**)&d_result, N * sizeof(hipDoubleComplex)), 
                  "Allocation résultat");
    
    // Copie des données vers le GPU
    checkCudaError(hipMemcpy(d_signal_real, signal_real, N * sizeof(double), 
                  hipMemcpyHostToDevice), "Copie signal réel vers GPU");
    checkCudaError(hipMemcpy(d_signal_imag, signal_imag, N * sizeof(double), 
                  hipMemcpyHostToDevice), "Copie signal imaginaire vers GPU");
    
    // Configuration de la grille et des blocs
    int numBlocks = (N + BLOCK_SIZE - 1) / BLOCK_SIZE;
    dim3 gridDim(numBlocks);
    dim3 blockDim(BLOCK_SIZE);
    
    // Lancement du kernel
    dftKernel<<<gridDim, blockDim>>>(d_signal_real, d_signal_imag, d_result, N);
    
    // Vérification des erreurs du kernel
    checkCudaError(hipGetLastError(), "Lancement du kernel");
    checkCudaError(hipDeviceSynchronize(), "Synchronisation du kernel");
    
    // Copie des résultats vers le CPU
    hipDoubleComplex* h_result = (hipDoubleComplex*)malloc(N * sizeof(hipDoubleComplex));
    checkCudaError(hipMemcpy(h_result, d_result, N * sizeof(hipDoubleComplex), 
                  hipMemcpyDeviceToHost), "Copie résultats vers CPU");
    
    // Conversion des résultats en format ComplexNumber
    for (int i = 0; i < N; i++) {
        result[i].real = h_result[i].x;
        result[i].imag = h_result[i].y;
    }
    
    // Libération de la mémoire
    free(signal_real);
    free(signal_imag);
    free(h_result);
    hipFree(d_signal_real);
    hipFree(d_signal_imag);
    hipFree(d_result);
}

int main() {
    int N = 10240;
    ComplexNumber* signal = (ComplexNumber*)malloc(N * sizeof(ComplexNumber));
    ComplexNumber* result = (ComplexNumber*)malloc(N * sizeof(ComplexNumber));
    
    if (signal == NULL || result == NULL) {
        printf("Erreur d'allocation mémoire.\n");
        return 1;
    }
    
    printf("Création du signal complexe...\n");
    for (int i = 0; i < N; i++) {
        signal[i].real = sin(2 * PI * 50 * i / N);
        signal[i].imag = cos(2 * PI * 120 * i / N);
    }
    
    printf("Calcul de la DFT avec CUDA...\n");
    struct timeval start, end;
    gettimeofday(&start, NULL);
    
    computeDFT_CUDA(signal, N, result);
    
    gettimeofday(&end, NULL);
    double time_spent = (end.tv_sec - start.tv_sec) * 1.0 + 
                       (end.tv_usec - start.tv_usec) / 1e6;
    
    printf("Temps d'exécution CUDA : %f secondes\n", time_spent);
    printf("Résultats de la DFT (partiels) :\n");
    for (int k = 0; k < 10; k++) {
        double magnitude = sqrt(result[k].real * result[k].real + 
                              result[k].imag * result[k].imag);
        double phase = atan2(result[k].imag, result[k].real);
        printf("k = %d : Magnitude = %.5f, Phase = %.5f radians\n", 
               k, magnitude, phase);
    }
    
    free(signal);
    free(result);
    
    return 0;
}